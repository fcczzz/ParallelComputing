#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <opencv2/opencv.hpp>

struct Point {
    int x0, y0;     //像素坐标
    double x, y, z; // RGB
    Point(int x0 = 0, int y0 = 0, int x = 0, int y = 0,
          int z = 0) :
        x0(x0),
        y0(y0), x(x), y(y), z(z) {
    }
    Point operator-(const Point &b) {
        return Point(0, 0, x - b.x, y - b.y, z - b.z);
    }
    Point operator+(const Point &b) {
        return Point(0, 0, x + b.x, y + b.y, z + b.z);
    }
    Point operator*(double k) {
        return Point(0, 0, x * k, y * k, z * k);
    }
    Point operator+=(const Point &b) {
        x += b.x;
        y += b.y;
        z += b.z;
        return *this;
    }
};

double dis(Point a, Point b) {
    return sqrt((a.x - b.x) * (a.x - b.x)
                + (a.y - b.y) * (a.y - b.y)
                + (a.z - b.z) * (a.z - b.z));
}

double Kernel(double x, double h) {
    //高斯核函数
    return exp(-x * x / (2 * h * h)) / (sqrt(2 * M_PI) * h);
}

int main() {
    cv::Mat image = cv::imread("input.jpg");

    int n = image.size().height, m = image.size().width;
    Point *src_array = new Point[n * m];
    Point *dst_array = new Point[n * m];

    int h = 10;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            cv::Vec3b pixel = image.at<cv::Vec3b>(i, j);
            src_array[i * m + j] =
                Point(i, j, pixel[0], pixel[1], pixel[2]);
        }
    }

    int N = n * m;
    for (int i = 0; i < N; i++) {
        Point p = src_array[i];
        Point nxt = p;
        if (i % 100 == 0)
            std::cout << i << " " << p.x0 << " " << p.y0
                      << " " << p.x << " " << p.y << " "
                      << p.z << std::endl;

        do {
            p = nxt;
            double sum = 0;
            for (int j = 0; j < N; j++) {
                Point delta = src_array[j] - p;

                double d = dis(p, src_array[j]);
                double w = Kernel(d, h);

                sum += w;
                nxt += delta * w;
            }
            nxt = nxt * (1 / sum);
            // std::cout << nxt.x << " " << nxt.y << " "
            //           << nxt.z << " " << dis(p, nxt)
            //           << std::endl;

        } while (dis(p, nxt) > 0.1);

        dst_array[i] = nxt;
    }

    cv::Mat dst_image(n, m, CV_8UC3);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            Point p = src_array[i * m + j];
            int x = p.x + 0.5, y = p.y + 0.5, z = p.z + 0.5;
            x = std::min(255, std::max(0, x));
            y = std::min(255, std::max(0, y));
            z = std::min(255, std::max(0, z));

            dst_image.at<cv::Vec3b>(i, j) =
                cv::Vec3b(p.x, p.y, p.z);
        }
    }

    cv::imwrite("output.jpg", dst_image);

    delete[] src_array;
    delete[] dst_array;
}